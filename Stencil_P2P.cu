#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstring>
#include <cstdlib>
#include <omp.h>
#include <utility>	//C++11
#include "time/seconds.h"

#define BLOCK 32
#define X 32
#define Y 32
#define Z 32
#define ELEM (size_t)(X*Y*Z)
#define STEP 32
#define GPUNUM 2
#define SLV (1*X*Y)

using namespace std;

#define CHECK(call)										\
{															\
	const hipError_t error = call;						\
	if(error != hipSuccess){								\
		cerr << "Error:" << __FILE__ << endl; \
		cerr << "code : "<< error << " reason : "<<hipGetErrorString(error) << endl;	\
	}														\
}


//P2P Functions
inline bool isCapableP2P(int ngpus)
{
    hipDeviceProp_t prop[ngpus];
    int iCount = 0;

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipGetDeviceProperties(&prop[i], i));

        if (prop[i].major >= 2) iCount++;

        printf("> GPU%d: %s %s capable of Peer-to-Peer access\n", i,
               prop[i].name, (prop[i].major >= 2 ? "is" : "not"));
    }

    if(iCount != ngpus)
    {
        printf("> no enough device to run this application\n");
    }

    return (iCount == ngpus);
}

inline void enableP2P (int ngpus)
{
    for( int i = 0; i < ngpus; i++ )
    {
        CHECK(hipSetDevice(i));

        for(int j = 0; j < ngpus; j++)
        {
            if(i == j) continue;

            int peer_access_available = 0;
            CHECK(hipDeviceCanAccessPeer(&peer_access_available, i, j));

            if (peer_access_available)
            {
                CHECK(hipDeviceEnablePeerAccess(j, 0));
                printf("> GPU%d enabled direct access to GPU%d\n", i, j);
            }
            else
            {
                printf("(%d, %d)\n", i, j );
            }
        }
    }
}

inline void disableP2P (int ngpus)
{
    for( int i = 0; i < ngpus; i++ )
    {
        CHECK(hipSetDevice(i));

        for(int j = 0; j < ngpus; j++)
        {
            if( i == j ) continue;

            int peer_access_available = 0;
            CHECK(hipDeviceCanAccessPeer( &peer_access_available, i, j) );

            if( peer_access_available )
            {
                CHECK(hipDeviceDisablePeerAccess(j));
                printf("> GPU%d disabled direct access to GPU%d\n", i, j);
            }
        }
    }
}


void checkResult(float* hostRef,float* devRef,const int N){
  float epsilon = 1e-3;
  bool match = 1;
  int i;
  float Ref;
  float Host;
  float Dev;
  for(i=0;i<N;i++){
  	Host = hostRef[i];
  	Dev = devRef[i];
  	Ref = Host-Dev;

    //printf("host:%d,device:%d\n",hostRef[i],devRef[i]);
    if((float)fabsf(Ref)>epsilon){
      match = 0;
      cout << "Arrays don't match.on count of "<<i<< " element." <<endl;
      cout << "Elapsed : " << Ref << " Host : " << Host << " | GPU : " << Dev << endl; 
      printf("Elapsed : %f Host : %f GPU : %f\n",Ref,Host,Dev );
      break;
    }
  }
  if(match){
    cout <<"Arrays match.";
  }
  cout << endl;
  return;
}

void initializeData(float* A,const int size){
  //乱数で値を初期化します。
  time_t t;
  int i;
  srand((unsigned int)time(&t));

  for(i=0;i<size;i++){
    A[i] = (float)(rand()&0xFFFF) / 0xFFFF;
  }
  return;
}
void print(float* Src){
	for(int i=0;i<ELEM;i++){
		cout << Src[i] << " ";
		if((i+1)%X==0)
			cout << endl;
	}
}
void print(float* Def,float* Src,float* Rst,const int elem){
	for(int i=0;i<elem;i++){
		cout << "\t" <<i << " | " << Def[i] << " | " <<Src[i] << " | "<<Rst[i] << endl;
	}
}

void Host3DStencil(float* Src,float* Dst){
	for(int time_step=0;time_step<STEP;time_step++){
		for(int all_loop=0;all_loop<ELEM;all_loop++){
			int mat_x = all_loop%X;
			int mat_y = all_loop/X;
			int mat_z = all_loop/(X*Y);

			if(mat_x!=0 && mat_x!=X-1 && mat_y!=0 && mat_y!= Y-1 && mat_z!=0 && mat_z!=Z-1){
				Dst[all_loop] = 0.4*Src[all_loop] + 0.1*(Src[all_loop-1]+Src[all_loop+1]+Src[all_loop-X]+Src[all_loop+X]+Src[all_loop-X*Y]+Src[all_loop+X*Y]);
			}
		}
		swap(Src,Dst);
		//cout << "iter : " << time_step << endl;
	}
}
 
//Multi版
__global__ void StencilOneStep(float* Src,float* Dst,const int MainElem,const int Dev){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	size_t mat_x = index % X;
	size_t mat_y = index / X;
	
	//デバイス番号によって動作が変わる
	switch(Dev){
		case 0:
			if(index>SLV+X*Y && index<MainElem+2*SLV-X*Y && mat_x != 0 && mat_x != X-1 && mat_y != 0 && mat_y != Y-1){
				Dst[index] = 0.4*Src[index] + 0.1*(Src[index-1]+Src[index+1]+Src[index+X]+Src[index-X]+Src[index+X*Y]+Src[index-X*Y]);
			}
			break;
		case GPUNUM-1:
			if(index>X*Y && index<MainElem+SLV-X*Y && mat_x != 0 && mat_x != X-1 && mat_y != 0 && mat_y != Y-1){
				Dst[index] = 0.4*Src[index] + 0.1*(Src[index-1]+Src[index+1]+Src[index+X]+Src[index-X]+Src[index+X*Y]+Src[index-X*Y]);
			}
			break;
		default:
			if(index>X*Y && index<MainElem+2*SLV-X*Y && mat_x != 0 && mat_x != X-1 && mat_y != 0 && mat_y != Y-1){
				Dst[index] = 0.4*Src[index] + 0.1*(Src[index-1]+Src[index+1]+Src[index+X]+Src[index-X]+Src[index+X*Y]+Src[index-X*Y]);
			}
			break;
	}
}

int main(int argc,char** argv){
	float* Src = new float[ELEM];
	float* Dst = new float[ELEM];
	float* Rst = new float[ELEM];
	float* Def = new float[ELEM];
	//Srcを乱数で初期化
	initializeData(Src,ELEM);

	memcpy(Dst,Src,sizeof(float)*ELEM);
	memcpy(Def,Src,sizeof(float)*ELEM);
	//HostTemp SLV
	float* Left = new float[SLV*GPUNUM];
	float* Right = new float[SLV*GPUNUM];

	omp_set_num_threads(GPUNUM);
	float **d_Src = new float*[GPUNUM];
	float **d_Dst = new float*[GPUNUM];

	//P2P
	isCapableP2P(GPUNUM);
	enableP2P(GPUNUM);
	double start,end;
	start = seconds()

	#pragma omp parallel
	{
		size_t MainElem = ELEM/GPUNUM;
		size_t CalcElem = MainElem + 2*SLV;
		size_t MainSize = MainElem * sizeof(float);
		size_t SleeveSize = SLV*sizeof(float);//Single
		size_t DeviceMemorySize = CalcElem * sizeof(float);

		//Device番号の取得
		int Dev = omp_get_thread_num();
		CHECK(hipSetDevice(Dev));
		
		 cout << Dev << " : MainElem -> " <<MainElem << " : CalcElem -> " <<CalcElem << " : SLV -> " << SLV << endl;
		//実行定義
		dim3 block(BLOCK);
		dim3 grid((CalcElem+block.x-1)/block.x);

		//開始のアドレス(要素番号)
		size_t MainAddress = Dev*MainElem;

		cout << Dev << " : StartAddress -> " << MainAddress << endl;

//		float *d_Src,*d_Dst;
		CHECK(hipMalloc(&d_Src[Dev],DeviceMemorySize));
		CHECK(hipMalloc(&d_Dst[Dev],DeviceMemorySize));
		//Init
		CHECK(hipMemset(d_Src[Dev],0,DeviceMemorySize));
		CHECK(hipMemset(d_Dst[Dev],0,DeviceMemorySize));

		//Memcpy
		if(Dev==0){
			// cout << "Copy GPU : " << Dev << " : Src ["<<MainAddress<<"]" << " : "<< MainSize+SleeveSize <<"Byte" <<endl;
			CHECK(hipMemcpy(&d_Src[Dev][SLV],&Src[MainAddress],MainSize+SleeveSize,hipMemcpyHostToDevice));
			CHECK(hipMemcpy(&d_Dst[Dev][SLV],&Src[MainAddress],MainSize+SleeveSize,hipMemcpyHostToDevice));
		}else if(Dev==GPUNUM-1){
			//cout << "Copy GPU : " << Dev << " : Src ["<<MainAddress<<"]" << " : "<< MainSize+SleeveSize <<"Byte" <<endl;
			CHECK(hipMemcpy(&d_Src[Dev][0],&Src[MainAddress-SLV],MainSize+SleeveSize,hipMemcpyHostToDevice));
			CHECK(hipMemcpy(&d_Dst[Dev][0],&Src[MainAddress-SLV],MainSize+SleeveSize,hipMemcpyHostToDevice));
		}else{
			//cout << "Copy GPU : " << Dev << " : Src ["<<MainAddress<<"]" << " : "<< MainSize+2*SleeveSize <<"Byte" <<endl;
			CHECK(hipMemcpy(&d_Src[Dev][0],&Src[MainAddress-SLV],MainSize+2*SleeveSize,hipMemcpyHostToDevice));
			CHECK(hipMemcpy(&d_Dst[Dev][0],&Src[MainAddress-SLV],MainSize+2*SleeveSize,hipMemcpyHostToDevice));
		}
//		cout << "block : "<< block.x << " | grid : " << grid.x << endl;
		for(int st=0;st<STEP;st++){
			//Stencil Calc
			//cout << Dev <<" : iter -> " << st << endl;
			StencilOneStep<<<grid,block>>>(d_Src[Dev],d_Dst[Dev],MainElem,Dev);
			swap(d_Src[Dev],d_Dst[Dev]);
			#pragma omp barrier
			if(Dev!=0){
				CHECK(hipMemcpy(&d_Src[Dev-1][SLV+MainElem],&d_Src[Dev][SLV],SleeveSize,hipMemcpyDeviceToDevice));
			}
			if(Dev!=GPUNUM-1){
				CHECK(hipMemcpy(&d_Src[Dev+1][0],&d_Src[Dev][MainElem],SleeveSize,hipMemcpyDeviceToDevice));
			}
			#pragma omp barrier
			CHECK(hipDeviceSynchronize());
			/*
			if(Dev!=0){
				CHECK(hipMemcpy(&d_Src[0],&Right[(Dev-1)*SLV],SleeveSize,hipMemcpyHostToDevice));
			}
			if(Dev!=GPUNUM-1){
				CHECK(hipMemcpy(&d_Src[SLV+MainElem],&Left[(Dev+1)*SLV],SleeveSize,hipMemcpyHostToDevice));
			}
			*/
		}

		CHECK(hipMemcpy(&Rst[Dev*MainElem],&d_Src[Dev][SLV],MainSize,hipMemcpyDeviceToHost));
		CHECK(hipGetLastError());

		CHECK(hipFree(d_Src[Dev]));
		CHECK(hipFree(d_Dst[Dev]));
	}
	end = seconds()

	disableP2P(GPUNUM);

	cout << "GPU Calc Finished." << endl;
	Host3DStencil(Src,Dst);
	cout << "CPU Calc Finished." << endl;
	checkResult(Src,Rst,ELEM);

	int elements = ELEM;
	int gpus = GPUNUM;
	int steps = STEPS;
	printf("------------------------------------------------\n");
	printf("Program : %s\n", argv[0]);
	printf("STEPS : %d\n", steps);
	printf("GPU : %d | ELEMENTS : %d  \n",gpus,elements );
	printf("Elapsed Time : %lf\n",end-start);
	printf("------------------------------------------------\n");


//	print(Def,Src,Rst,ELEM);
	delete Src;
	delete Dst;
	delete Rst;
	delete Def;
	return 0;
}