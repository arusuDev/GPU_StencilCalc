//2次元配列における複数台GPUを用いたステンシル計算

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cstring>
#include <cstdlib>
#include <utility>	//C++11
#include <omp.h>


#define BLOCK 32
#define GPUNUM 2
#define X 32
#define Y 32
#define ELEM (size_t)(X*Y)
#define STEP 100
#define SLV (1*X)
using namespace std;

#define CHECK(call)										\
{															\
	const hipError_t error = call;						\
	if(error != hipSuccess){								\
		cerr << "Error:" << __FILE__ << endl; \
		cerr << "code : "<< error << " reason : "<<hipGetErrorString(error) << endl;	\
	}														\
}


void checkResult(float* hostRef,float* devRef,const int N){
  float epsilon = 1e-3;
  bool match = 1;
  int i;
  float Ref;
  float Host;
  float Dev;
  for(i=0;i<N;i++){
  	Host = hostRef[i];
  	Dev = devRef[i];
  	Ref = Host-Dev;

    //printf("host:%d,device:%d\n",hostRef[i],devRef[i]);
    if((float)fabsf(Ref)>epsilon){
      match = 0;
      cout << "Arrays don't match.on count of "<<i<< " element." <<endl;
      cout << "Elapsed : " << Ref << " Host : " << Host << " | GPU : " << Dev << endl; 
      printf("Elapsed : %f Host : %f GPU : %f\n",Ref,Host,Dev );
      break;
    }
  }
  if(match){
    cout <<"Arrays match.";
  }
  cout << endl;
  return;
}

void initializeData(float* A,int size){
  //乱数で値を初期化します。
  time_t t;
  int i;
  srand((unsigned int)time(&t));

  for(i=0;i<size;i++){
    A[i] = (float)(rand()*0xFFF) / 10000000.0F;
  }
  return;
}
void print(float* Src){
	for(int i=0;i<ELEM;i++){
		cout << Src[i] << " ";
		if((i+1)%X==0)
			cout << endl;
	}
}
void print(float* Def,float* Src,float* Rst,const int elem){
	for(int i=0;i<elem;i++){
		cout << "\t" <<i << " | " << Def[i] << " | " <<Src[i] << " | "<<Rst[i] << endl;
	}
}

void Host2DStencil(float* Src,float* Dst){
	for(int time_step=0;time_step<STEP;time_step++){
		for(size_t all_loop=0;all_loop<ELEM;all_loop++){
			int mat_x=all_loop%X;//X成分
			int mat_y=all_loop/X;//Y成分
			//cout << "Time: "<<time_step<< " | X:" << mat_x << " | Y:"<<mat_y ;
			if(mat_x!=0 && mat_x!=X-1 && mat_y!=0 && mat_y!=Y-1){
				//端ならば計算しない
				Dst[all_loop] = 0.6*Src[all_loop] + 0.1*(Src[all_loop-1]+Src[all_loop+1]+Src[all_loop-X]+Src[all_loop+X]);
			}
		}
		swap(Src,Dst);
	}
}
//Single版
/*
__global__ void StencilOneStep(float* Src,float* Dst){
	size_t  index = threadIdx.x + blockDim.x * blockIdx.x;
	size_t mat_x = index % X; //X成分
	size_t mat_y = index / X; //Y成分
	if(mat_x != 0 && mat_x != X-1 && mat_y != 0 && mat_y != Y-1){
		Dst[index] = 0.6*Src[index] + 0.1*(Src[index-1] + Src[index+1] + Src[index+X] + Src[index-X]);
	}
} 
*/
//Multi版
__global__ void StencilOneStep(float* Src,float* Dst,const int MainElem,const int Dev){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	size_t mat_x = index % X;
	
	//デバイス番号によって動作が変わる
	switch(Dev){
		case 0:
			if(index>SLV+X && index<MainElem+2*SLV-X && mat_x != 0 && mat_x != X-1){
				Dst[index] = 0.6*Src[index] + 0.1*(Src[index-1]+Src[index+1]+Src[index+X]+Src[index-X]);
			}
			break;
		case GPUNUM-1:
			if(index>X && index<MainElem+SLV-X && mat_x != 0 && mat_x != X-1){
				Dst[index] = 0.6*Src[index] + 0.1*(Src[index-1]+Src[index+1]+Src[index+X]+Src[index-X]);
			}
			break;
		default:
			if(index>X && index<MainElem+2*SLV-X&& mat_x != 0 && mat_x != X-1){
				Dst[index] = 0.6*Src[index] + 0.1*(Src[index-1]+Src[index+1]+Src[index+X]+Src[index-X]);
			}
			break;
	}

}

int main(int argc,char** argv){
	float* Src = new float[ELEM];
	float* Dst = new float[ELEM];
	float* Rst = new float[ELEM];
	float* Def = new float[ELEM];
	//Srcを乱数で初期化
	initializeData(Src,ELEM);
	
	memcpy(Dst,Src,sizeof(float)*ELEM);
	memcpy(Def,Src,sizeof(float)*ELEM);

	//HostTemp SLV
	float* Left = new float[SLV*GPUNUM];
	float* Right = new float[SLV*GPUNUM];

	omp_set_num_threads(GPUNUM);
	//Deviceメモリの確保
	#pragma omp parallel
	{
		size_t MainElem = ELEM/GPUNUM;
		size_t CalcElem = MainElem + 2*SLV;
		size_t MainSize = MainElem * sizeof(float);
		size_t SleeveSize = SLV*sizeof(float);//Single
		size_t DeviceMemorySize = CalcElem * sizeof(float);

		//Device番号の取得
		int Dev = omp_get_thread_num();
		CHECK(hipSetDevice(Dev));
		
		// cout << Dev << " : MainElem -> " <<MainElem << " : CalcElem -> " <<CalcElem << " : SLV -> " << SLV << endl;
		//実行定義
		dim3 block(BLOCK);
		dim3 grid((CalcElem+block.x-1)/block.x);

		//開始のアドレス(要素番号)
		size_t MainAddress = Dev*MainElem;

		// cout << Dev << " : StartAddress -> " << MainAddress << endl;

		float *d_Src,*d_Dst;
		CHECK(hipMalloc(&d_Src,DeviceMemorySize));
		CHECK(hipMalloc(&d_Dst,DeviceMemorySize));
		//Init
		CHECK(hipMemset(d_Src,0,DeviceMemorySize));
		CHECK(hipMemset(d_Dst,0,DeviceMemorySize));

		//Memcpy
		if(Dev==0){
			// cout << "Copy GPU : " << Dev << " : Src ["<<MainAddress<<"]" << " : "<< MainSize+SleeveSize <<"Byte" <<endl;
			CHECK(hipMemcpy(&d_Src[SLV],&Src[MainAddress],MainSize+SleeveSize,hipMemcpyHostToDevice));
			CHECK(hipMemcpy(&d_Dst[SLV],&Src[MainAddress],MainSize+SleeveSize,hipMemcpyHostToDevice));
		}else if(Dev==GPUNUM-1){
			//cout << "Copy GPU : " << Dev << " : Src ["<<MainAddress<<"]" << " : "<< MainSize+SleeveSize <<"Byte" <<endl;
			CHECK(hipMemcpy(&d_Src[0],&Src[MainAddress-SLV],MainSize+SleeveSize,hipMemcpyHostToDevice));
			CHECK(hipMemcpy(&d_Dst[0],&Src[MainAddress-SLV],MainSize+SleeveSize,hipMemcpyHostToDevice));
		}else{
			//cout << "Copy GPU : " << Dev << " : Src ["<<MainAddress<<"]" << " : "<< MainSize+2*SleeveSize <<"Byte" <<endl;
			CHECK(hipMemcpy(&d_Src[0],&Src[MainAddress-SLV],MainSize+2*SleeveSize,hipMemcpyHostToDevice));
			CHECK(hipMemcpy(&d_Dst[0],&Src[MainAddress-SLV],MainSize+2*SleeveSize,hipMemcpyHostToDevice));
		}
//		cout << "block : "<< block.x << " | grid : " << grid.x << endl;
		for(int st=0;st<STEP;st++){
			//Stencil Calc
			//cout << Dev <<" : iter -> " << st << endl;
			StencilOneStep<<<grid,block>>>(d_Src,d_Dst,MainElem,Dev);
			swap(d_Src,d_Dst);
			if(Dev!=0){
				CHECK(hipMemcpy(&Left[Dev*SLV],&d_Src[SLV],SleeveSize,hipMemcpyDeviceToHost));
			}
			if(Dev!=GPUNUM-1){
				CHECK(hipMemcpy(&Right[Dev*SLV],&d_Src[MainElem],SleeveSize,hipMemcpyDeviceToHost));
			}

			#pragma omp barrier

			if(Dev!=0){
				CHECK(hipMemcpy(&d_Src[0],&Right[(Dev-1)*SLV],SleeveSize,hipMemcpyHostToDevice));
			}
			if(Dev!=GPUNUM-1){
				CHECK(hipMemcpy(&d_Src[SLV+MainElem],&Left[(Dev+1)*SLV],SleeveSize,hipMemcpyHostToDevice));
			}
		}

		CHECK(hipMemcpy(&Rst[Dev*MainElem],&d_Src[SLV],MainSize,hipMemcpyDeviceToHost));
		CHECK(hipGetLastError());

		CHECK(hipFree(d_Src));
		CHECK(hipFree(d_Dst));
	}

	Host2DStencil(Src,Dst);
	checkResult(Src,Rst,ELEM);

	delete Right;
	delete Left;

	delete Src;
	delete Dst;
	delete Rst;
	delete Def;
	return 0;
}